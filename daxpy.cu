/*
 * =====================================================================================
 *
 *       Filename:  daxpy.c
 *
 *    Description:  Test cublas DAXPY, specifically to verify usage on
 *                  summit with GPUMPS and all 6 GPUs shared over 42 procs.
 *
 *        Version:  1.0
 *        Created:  05/20/2019 10:33:30 AM
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *   Organization:  
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <stdlib.h>

#include "hipblas.h"
#include "hip/hip_runtime_api.h"

#define GPU_CHECK_CALLS
#include "cuda_error.h"

// column major
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static hipblasHandle_t handle;


int main(int argc, char **argv) {
    int n = 1024;

    double a = 2.0;
    double sum = 0.0;

    double *x, *y, *d_x, *d_y;

    x = (double *)malloc(n*sizeof(*x));
    if (x == NULL) {
        printf("host malloc(x) failed\n");
        return EXIT_FAILURE;
    }

    y = (double *)malloc(n*sizeof(*y));
    if (x == NULL) {
        printf("host malloc(y) failed\n");
        return EXIT_FAILURE;
    }

    for (int i=0; i<n; i++) {
        x[i] = i+1;
        y[i] = -i-1;
    }

    //CHECK("setDevice", hipSetDevice(0));

    CHECK( "cublas", hipblasCreate(&handle) );

    CHECK( "d_x", hipMalloc((void**)&d_x, n*sizeof(*d_x)) );
    CHECK( "d_y", hipMalloc((void**)&d_y, n*sizeof(*d_y)) );

    CHECK("d_x = x",
          hipMemcpy(d_x, x, n*sizeof(*x), hipMemcpyHostToDevice) );
    CHECK("d_y = y",
          hipMemcpy(d_y, y, n*sizeof(*y), hipMemcpyHostToDevice) );

    CHECK("daxpy",
          hipblasDaxpy(handle, n, &a, d_x, 1, d_y, 1) );

    CHECK("daxpy sync", hipDeviceSynchronize());
    
    CHECK("y = d_y",
          hipMemcpy(y, d_y, n*sizeof(*y), hipMemcpyDeviceToHost) );

    CHECK("y = d_y sync", hipDeviceSynchronize() );

    sum = 0.0;
    for (int i=0; i<n; i++) {
        printf("%f\n", y[i]);
        sum += y[i];
    }
    printf("SUM = %f\n", sum);

    // cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);
    return EXIT_SUCCESS;
}
